/*****************************************************
* This file tests cuda memory management APIs.
*****************************************************/
#include <hip/hip_runtime.h>
#include <stdio.h>

void initialData(float *h, long long n, float data) {
    long long i;
    for (i = 0; i < n; i++) {
        h[i] = data;
    }
}

__global__ void vecAdd(float* A, float* B, float* C)
{
    long long i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

void test_cudaDeviceGetAttribute(int* value, hipDeviceAttribute_t attr, int device) {
    hipDeviceGetAttribute(value, attr, device);
}

void test_cudaMallocManaged(int dev, int ipower) {
    int val;

    // Check if supports managed memory
    hipDeviceGetAttribute(&val, hipDeviceAttributeManagedMemory, dev);
    if (!val) {
        printf("*** Error: Managed memory is not supported!\n");
        return;
    }

    // Check concurrent managed access, for cuda 8.0
    hipDeviceGetAttribute(&val, hipDeviceAttributeConcurrentManagedAccess, dev);
    if (!val) {
        printf("*** Warn: Concurrent managed access is not supported!\n");
    }

    // Calculate number of elements and bytes
    long long nElem = ((long long)1)<<ipower;
    long long nBytes = nElem * sizeof(float);
    if (ipower < 18) {
        printf("Vector size is %lld, nbytes is %f KB\n", nElem, (float)nBytes/(1024.0f));
    } else {
        printf("Vector size is %lld, nbytes is %f MB\n", nElem, (float)nBytes/(1024.0f*1024.0f));
    }

    // allocate memory
    float *g_A, *g_B, *g_C;
    float *g_D, *g_E, *g_F;
    // unsigned int flags = cudaMemAttachHost;
    unsigned int flags = hipMemAttachGlobal;
    int ret1 = hipMallocManaged(&g_A, nBytes, flags);
    int ret2 = hipMallocManaged(&g_B, nBytes, flags);
    int ret3 = hipMallocManaged(&g_C, nBytes, flags);
    int ret4 = hipMallocManaged(&g_D, nBytes, flags);
    int ret5 = hipMallocManaged(&g_E, nBytes, flags);
    int ret6 = hipMallocManaged(&g_F, nBytes, flags);
    printf("===== %d %d %d %d %d %d\n", ret1, ret2, ret3, ret4, ret5, ret6);

    printf("===== inital data begins...\n");
    initialData(g_A, nElem, 2.0f);
    initialData(g_B, nElem, 2.0f);
    initialData(g_D, nElem, 2.0f);
    initialData(g_E, nElem, 2.0f);
    printf("===== synchronize begins...\n");
    hipDeviceSynchronize();

    printf("===== add data begins...\n");
    dim3 threadsPerBlock(1024);
    dim3 numBlocks((nElem)/threadsPerBlock.x);
    printf("===== numBlocks is %d, threadsPerBlock is %d\n", numBlocks.x, threadsPerBlock.x);
    // Kernel invocation with N threads
    vecAdd<<<numBlocks, threadsPerBlock>>>(g_A, g_B, g_C);
    hipDeviceSynchronize();
    vecAdd<<<numBlocks, threadsPerBlock>>>(g_D, g_E, g_F);
    hipDeviceSynchronize();

    printf("===== check the results...\n");

    long long i;
    float ans = 4.0f;
    printf("===== ans is %f\n", ans);
    for (i = 0; i < nElem; i++) {
        if (g_C[i] != ans) {
            printf("Result g_C[%lld] is error, error value is %3.0f\n", i, g_C[i]);
            break;
        }
        if (g_F[i] != ans) {
            printf("Result g_F[%lld] is error, error value is %3.0f\n", i, g_F[i]);
            break;
        }
        // printf("Result g_C[%lld] value is %3.0f\n", i, g_C[i]);
    }

    hipFree(g_A);
    hipFree(g_B);
    hipFree(g_C);
    hipFree(g_D);
    hipFree(g_E);
    hipFree(g_F);
    hipDeviceReset();

}

int main(int argc, char* argv[]) {

    // set up device
    int dev = 0;
    hipSetDevice(dev);

    // get device properties
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    // check uva supporting
    if (deviceProp.unifiedAddressing) {
        printf("Device %d supports uva memory!\n", dev);
    }
    else {
        printf("Device %d does not support uva memory!\n", dev);
        exit(EXIT_SUCCESS);
    }

    // set up date size of vectors
    int ipower = 10;
    if (argc > 1) ipower = atoi(argv[1]);

    test_cudaMallocManaged(dev, ipower);

}
